#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
int pointers()
{
	int* a;
	hipMalloc(&a, 100);
	hipFree(a);

	int test = 10;
	int* pt = &test;

	std::cout << "this is the pointer: " << pt << std::endl;
	std::cout << "this is the value: " << *pt << std::endl;


	int value = 42;
	int* ptr1 = &value;
	int** ptr2 = &ptr1;
	int*** ptr3 = &ptr2;

	std::cout << "value = " << ***ptr3 << std::endl;
	std::cout << "pointer = " << ptr3 << std::endl;

	int num = 10;
	float fnum = 3.14;
	void* vptr;

	vptr = &fnum;
	std::cout << "vptr = " << vptr << std::endl;



	return 0;
}