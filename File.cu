#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cassert>
#include <iostream>
#include <cstdlib>

using namespace std;

__global__ void matrixMultiplication(int* a, int* b, int* c, int N)
{
	//calc global row & column for each thread
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check for matrix
	if (row < N && col < N) {
		int temp = 0;
		for (int i = 0; i < N; i++) {
			temp += a[row * N + i] * b[i * N + col];
		}
		//write results
		c[row * N + col] = temp;
	}
}

//init square matrix w random nums

void init_matrix(int* m, int N) {
	for (int i = 0; i < N * N; i++) {
		m[i] = rand() % 100;
	}
}

//verify result on cpu
void verify_result(int* a, int* b, int* c, int N) {
	int temp;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			temp = 0;
			for (int k = 0; k < N; k++) {
				temp += a[i * N + k] * b[k * N + j];
			}
			//check each result
			assert(temp = c[i * N + j]);
		}
	}
}

int main() {
	//set square matrix dimension
	int N = 1 << 10;
	size_t bytes = N * N * sizeof(int);

	//allocate memory for matrices
	int* a, * b, * c;
	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);

	// init matrices w/ random nums
	init_matrix(a, N);
	init_matrix(b, N);

	//set cooperative thread array and grid dimensions
	int threads = 16;
	int blocks = (N + threads - 1) / threads;

	// setup kernel launch params
	dim3 THREADS(threads, threads);
	dim3 BLOCKS(blocks, blocks);

	// launch kernel
	matrixMultiplication << <BLOCKS, THREADS >> > (a, b, c, N);
	hipDeviceSynchronize();

	//verify result
	//verify_result(a, b, c, N);



	cout << "PROGRAM COMPLETED SUCCESSFULLY" << endl;
	return 0;
}
